// Copyright 2004-, Facebook, Inc. All Rights Reserved.

#include <stdio.h>
#include <stdexcept>
#include <hip/hip_runtime.h>

#include "src/HalfPrec.h"
#include "src/util/Transform.cuh"

using namespace facebook::cuda;
void halfprec_ToHalf(hipStream_t stream,
                     const float* input,
                     half_t* output,
                     size_t n) {
  transform<ToHalf>(stream, input, output, n);
}

void halfprec_ToFloat(hipStream_t stream,
                      const half_t* input,
                      float* output,
                      size_t n) {
  transform<ToFloat>(stream, input, output, n);
}
